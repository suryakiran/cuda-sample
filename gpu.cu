#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/scatter.h>
#include "gpu.hpp"

void
printCudaVersion()
{
    int runtime_ver;
    hipRuntimeGetVersion(&runtime_ver);
    std::cout << "CUDA Runtime version: " << runtime_ver << std::endl;

    int driver_ver;
    hipDriverGetVersion(&driver_ver);
    std::cout << "CUDA Driver version: " << driver_ver << std::endl;
}

void
thrustTest()
{
    std::cout << " ***** Thrust Test begin *****" << std::endl;
    // mark even indices with a 1; odd indices with a 0
    int                        values[10] = {1, 2, 3, 4, 5, 6, 7, 8, 9, 10};
    thrust::device_vector<int> d_values(values, values + 10);

    // scatter all even indices into the first half of the
    // range, and odd indices vice versa
    int                        map[10] = {0, 5, 1, 6, 2, 7, 3, 8, 4, 9};
    thrust::device_vector<int> d_map(map, map + 10);
    thrust::device_vector<int> d_output(10);
    thrust::scatter(thrust::device, d_values.begin(), d_values.end(), d_map.begin(), d_output.begin());

    thrust::host_vector<int> out(d_output);
    for (auto item : out) {
        std::cout << item << '\t';
    }
    std::cout << std::endl;
    std::cout << " ***** Thrust Test end *****" << std::endl;
}
