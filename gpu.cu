#include <iostream>
#include <numeric>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/scatter.h>
#include <boost/range/algorithm_ext.hpp>
#include "gpu.hpp"

void
printCudaVersion()
{
    int runtime_ver;
    hipRuntimeGetVersion(&runtime_ver);
    std::cout << "CUDA Runtime version: " << runtime_ver << std::endl;

    int driver_ver;
    hipDriverGetVersion(&driver_ver);
    std::cout << "CUDA Driver version: " << driver_ver << std::endl;
}

void
thrustTest()
{
    std::cout << " ***** Thrust Test begin *****" << std::endl;
    // mark even indices with a 1; odd indices with a 0
    int                        values[10] = {1, 2, 3, 4, 5, 6, 7, 8, 9, 10};
    thrust::device_vector<int> d_values(values, values + 10);

    // scatter all even indices into the first half of the
    // range, and odd indices vice versa
    int                        map[10] = {0, 5, 1, 6, 2, 7, 3, 8, 4, 9};
    thrust::device_vector<int> d_map(map, map + 10);
    thrust::device_vector<int> d_output(10);
    thrust::scatter(thrust::device, d_values.begin(), d_values.end(), d_map.begin(), d_output.begin());

    thrust::host_vector<int> out(d_output);
    for (auto item : out) {
        std::cout << item << '\t';
    }
    std::cout << std::endl;
    std::cout << " ***** Thrust Test end *****" << std::endl;
}

void
getNewIndices(const std::vector<int>& input, size_t xs, size_t ys, size_t kernel)
{
    size_t nx{xs / kernel}, ny{ys / kernel};

    std::cout << nx << '\t' << ny << std::endl;
}

void
imageTest()
{
    size_t xs(8), ys(8), kernel(2);

    std::vector<int> vi(xs * ys);
    boost::iota(vi, 0);

    getNewIndices(vi, xs, ys, kernel);
}