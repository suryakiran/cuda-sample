#include <iostream>
#include "gpu.hpp"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

void printCudaVersion()
{
    int runtime_ver;
    hipRuntimeGetVersion(&runtime_ver);
    std::cout << "CUDA Runtime version: " << runtime_ver << std::endl;

    int driver_ver;
    hipDriverGetVersion(&driver_ver);
    std::cout << "CUDA Driver version: " << driver_ver << std::endl;
}
