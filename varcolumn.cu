#include "varcolumn.h"

#include <thrust/device_ptr.h>
#include <thrust/fill.h>
#include <boost/core/demangle.hpp>
#include <boost/current_function.hpp>

void
varcolumn::fillData(double val)
{
    thrust::device_ptr<double> dptr(static_cast<double*>(data.get()));
    thrust::fill(dptr, dptr + elemCount, val);
}

double
varcolumn::print()
{
    double num{0};
    std::cout << BOOST_CURRENT_FUNCTION << std::endl;
    if (data) {
        hipSetDevice(0);

        thrust::device_ptr<double> dptr(reinterpret_cast<double*>(data.get()));
        return dptr[5];
    } else {
        std::cout << "No Data to print" << std::endl;
    }
    return num;
}
